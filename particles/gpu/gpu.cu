#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

void compute_grid_offset(int offset[9], int num_bins, int bin_size)
{
    offset[0] = -num_bins * bin_size - bin_size;
    offset[1] = -num_bins * bin_size;
    offset[2] = -num_bins * bin_size + bin_size;
    offset[3] = -bin_size;
    offset[4] = 0;
    offset[5] = bin_size;
    offset[6] = num_bins * bin_size - bin_size;
    offset[7] = num_bins * bin_size;
    offset[8] = num_bins * bin_size + bin_size;
}

void compute_bin_offset(int offset[9], int num_bins)
{
    offset[0] = -num_bins - 1;
    offset[1] = -num_bins;
    offset[2] = -num_bins + 1;
    offset[3] = -1;
    offset[4] = 0;
    offset[5] = 1;
    offset[6] = num_bins - 1;
    offset[7] = num_bins;
    offset[8] = num_bins + 1;
}

int grid_size()
{
    return (int)ceil(size/(2*cutoff)) + 2;
}

__device__ int get_bin_index(particle_t& p, int num_bins)
{
    int i = p.x / (2 * cutoff) + 1;
    int j = p.y / (2 * cutoff) + 1;
    return i * num_bins + j;
}

__device__ int get_grid_index(particle_t& p, int num_bins, int bin_size)
{
    int i = p.x / (2 * cutoff) + 1;
    int j = p.y / (2 * cutoff) + 1;
    return i * num_bins * bin_size + j * bin_size;
}

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void clear_bins(unsigned int* bin_sizes, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) {
        return;
    }
    
    bin_sizes[tid] = 0;
}

__global__ void bin_particles(particle_t* grid, particle_t* particles, unsigned int* bin_sizes, int n, int num_bins, int bin_size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) {
        return;
    }

    int bindex = get_bin_index(particles[tid], num_bins);
    int gindex = get_grid_index(particles[tid], num_bins, bin_size);

    int pos = atomicAdd(&bin_sizes[bindex], (unsigned int)1);
    grid[gindex + pos] = particles[tid];
}

__global__ void compute_forces_gpu(particle_t* grid, particle_t* particles, unsigned int* bin_sizes, int* goffset, int* boffset, int n, int num_bins, int bin_size)
{
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) {
        return;
    }

    particles[tid].ax = particles[tid].ay = 0;

    int bindex = get_bin_index(particles[tid], num_bins);
    int gindex = get_grid_index(particles[tid], num_bins, bin_size);

    for(int i = 0; i < 9; i++) {
        int gdx = gindex + goffset[i];
        int bdx = bindex + boffset[i];
        for(int j = 0; j < bin_sizes[bdx]; j++) {
            apply_force_gpu(particles[tid], grid[gdx + j]);
        }
    }
}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) { 
        return;
    }

    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char* savename = read_string( argc, argv, "-o", NULL );
    
    FILE* fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t* particles = (particle_t*) malloc( n * sizeof(particle_t) );
    particle_t* d_particles;
    particle_t* grid;
    unsigned int* bin_sizes;
    int* goffset;
    int* boffset;

    int offset[9];
    int num_bins;
    int bin_size = 10;
    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int clear_blks;

    set_size(n);
    init_particles(n, particles);

    num_bins = grid_size();    
    clear_blks = (num_bins*num_bins + NUM_THREADS - 1) / NUM_THREADS;

    hipMalloc((void **) &d_particles, n * sizeof(particle_t));
    hipMalloc((void **) &grid, num_bins * num_bins * bin_size * sizeof(particle_t));
    hipMalloc((void **) &bin_sizes, num_bins * num_bins * sizeof(unsigned int));
    hipMalloc((void **) &goffset, 9 * sizeof(int));
    hipMalloc((void **) &boffset, 9 * sizeof(int));    
 
    compute_grid_offset(offset, num_bins, bin_size);
    hipMemcpy(goffset, offset, 9 * sizeof(int), hipMemcpyHostToDevice);
    compute_bin_offset(offset, num_bins);
    hipMemcpy(boffset, offset, 9 * sizeof(int), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;

    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for(int step = 0; step < NSTEPS; step++) {

        clear_bins <<< clear_blks, NUM_THREADS >>> (bin_sizes, num_bins * num_bins);

        bin_particles <<< blks, NUM_THREADS >>> (grid, d_particles, bin_sizes, n, num_bins, bin_size);

        compute_forces_gpu <<< blks, NUM_THREADS >>> (grid, d_particles, bin_sizes, goffset, boffset, n, num_bins, bin_size);

        move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save(fsave, n, particles);
	    }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
   
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    hipFree(grid);
    hipFree(bin_sizes);
    hipFree(goffset);
    hipFree(boffset);
    if(fsave) {
        fclose(fsave);
    }
    return 0;
}
